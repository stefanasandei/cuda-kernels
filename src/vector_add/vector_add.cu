#include "hip/hip_runtime.h"
#include <common/common.h>
#include <common/cuda_common.h>

#include <hip/hip_runtime.h>
#include <>

__global__ void vectorAdd(int* a, int* b, int* c, int size)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    c[idx] = a[idx] + b[idx];
  }
}

void vectorAddHost(
    const int* h_a,
    const int* h_b,
    int* h_c,
    int size,
    BenchmarkOptional benchmarkOpt)
{
  int* d_a = nullptr;
  int* d_b = nullptr;
  int* d_c = nullptr;

  hipMalloc(&d_a, size * sizeof(int));
  hipMalloc(&d_b, size * sizeof(int));
  hipMalloc(&d_c, size * sizeof(int));

  hipMemcpy(d_a, h_a, size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size * sizeof(int), hipMemcpyHostToDevice);

  int threadsPerBlock = 256;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
  BenchmarkOrRun(
      benchmarkOpt,
      vectorAdd,
      blocksPerGrid,
      threadsPerBlock,

      d_a,
      d_b,
      d_c,
      size);

  hipMemcpy(h_c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
