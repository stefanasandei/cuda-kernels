#include "hip/hip_runtime.h"
#define ARR_LEN(x) (sizeof(x) / sizeof(int))

#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>

#include <common.h>

__global__ void vectorAdd(int* a, int* b, int* c)
{
  int idx = threadIdx.x;

  c[idx] = a[idx] + b[idx];
}

int vec()
{
  int a[] = {1, 2, 3};
  int b[] = {4, 5, 6};
  int c[ARR_LEN(a)] = {0};

  int* cudaA = nullptr;
  int* cudaB = nullptr;
  int* cudaC = nullptr;

  hipMalloc(&cudaA, sizeof(a));
  hipMalloc(&cudaB, sizeof(b));
  hipMalloc(&cudaC, sizeof(c));

  hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
  hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);
  hipMemcpy(cudaC, c, sizeof(c), hipMemcpyHostToDevice);

  vectorAdd<<<1, ARR_LEN(c)>>>(cudaA, cudaB, cudaC);

  hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

  for (const int i : c) {
    printf("%d ", i);
  }
  printf("\n");

  return 0;
}
