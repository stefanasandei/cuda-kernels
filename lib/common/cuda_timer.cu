//
// Created by stefan on 8/31/25.
//

#include "cuda_timer.h"

#include <hip/hip_runtime.h>
#include <memory>

struct CudaTimer::Impl
{
  hipEvent_t start_, stop_;
};

CudaTimer::CudaTimer() : pImpl_(std::make_unique<Impl>()), m_TimeMS(0)
{
  hipEventCreate(&pImpl_->start_);
  hipEventCreate(&pImpl_->stop_);
}

CudaTimer::~CudaTimer()
{
  hipEventDestroy(pImpl_->start_);
  hipEventDestroy(pImpl_->stop_);
}

void CudaTimer::start() const
{
  hipEventRecord(pImpl_->start_);
}

float CudaTimer::stop()
{
  hipEventRecord(pImpl_->stop_);
  hipEventSynchronize(pImpl_->stop_);
  float ms = 0.0f;
  hipEventElapsedTime(&ms, pImpl_->start_, pImpl_->stop_);

  m_TimeMS = ms;
  return ms;
}