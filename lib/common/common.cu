//
// Created by stefan on 8/31/25.
//

#include "common.h"

#include <stdexcept>

CudaTimer::CudaTimer()
{
  hipEventCreate(&start_);
  hipEventCreate(&stop_);
}
CudaTimer::~CudaTimer()
{
  hipEventDestroy(start_);
  hipEventDestroy(stop_);
}

void CudaTimer::start()
{
  hipEventRecord(start_);
}
float CudaTimer::stop()
{
  hipEventRecord(stop_);
  hipEventSynchronize(stop_);
  float ms = 0.0f;
  hipEventElapsedTime(&ms, start_, stop_);
  return ms;
}