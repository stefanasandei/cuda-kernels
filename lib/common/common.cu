//
// Created by stefan on 8/31/25.
//

#include "common.h"

#include <stdexcept>

CudaTimer::CudaTimer()
{
  hipEventCreate(&start_);
  hipEventCreate(&stop_);
}
CudaTimer::~CudaTimer()
{
  hipEventDestroy(start_);
  hipEventDestroy(stop_);
}

void CudaTimer::start() const
{
  hipEventRecord(start_);
}
float CudaTimer::stop() const
{
  hipEventRecord(stop_);
  hipEventSynchronize(stop_);
  float ms = 0.0f;
  hipEventElapsedTime(&ms, start_, stop_);
  return ms;
}