//
// Created by stefan on 8/31/25.
//

#include "common.h"
#include <hip/hip_runtime.h>
#include <memory>

struct CudaTimer::Impl
{
  hipEvent_t start_, stop_;
};

CudaTimer::CudaTimer() : pImpl_(std::make_unique<Impl>())
{
  hipEventCreate(&pImpl_->start_);
  hipEventCreate(&pImpl_->stop_);
}

CudaTimer::~CudaTimer()
{
  hipEventDestroy(pImpl_->start_);
  hipEventDestroy(pImpl_->stop_);
}

void CudaTimer::start() const
{
  hipEventRecord(pImpl_->start_);
}

float CudaTimer::stop() const
{
  hipEventRecord(pImpl_->stop_);
  hipEventSynchronize(pImpl_->stop_);
  float ms = 0.0f;
  hipEventElapsedTime(&ms, pImpl_->start_, pImpl_->stop_);
  return ms;
}